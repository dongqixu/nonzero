#include "hip/hip_runtime.h"
#define HIPCUB_STDERR

#include <hipcub/hipcub.hpp>
#include <stdio.h>
#include <iostream>
#include "nonzero.h"

const int RUNTIME_BLOCK_THREADS = 8;

template<typename T>
__host__ __device__ __forceinline__ T ATenCeilDiv(T a, T b) {
    return (a + b - 1) / b;
}

// sum number of nonzero elements in each block with reduce
template<typename T, int BLOCK_THREADS>
__global__ void nonZeroCountKernel(const T *inputs, int64_t length, int *counts) {
    typedef hipcub::BlockReduce<int, BLOCK_THREADS, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> BlockReduceT;
    __shared__ typename BlockReduceT::TempStorage temp_storage;
    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    hipcub::CastOp<bool> cast_to_bool;
    int num = 0;
    if (index < length && cast_to_bool(inputs[index])) {
        num++;
    }
    int aggregate = BlockReduceT(temp_storage).Sum(num);
    // record aggregated count number by thread 0
    if (threadIdx.x == 0) {
        counts[blockIdx.x] = aggregate;
    }
}

template<typename T, int BLOCK_THREADS>
__global__ void nonZeroIndexKernel(const T *inputs, int64_t length, const int *cumulativeCounts, int *output) {
    typedef hipcub::BlockScan<int, BLOCK_THREADS, hipcub::BLOCK_SCAN_RAKING> BlockScanT;
    __shared__ typename BlockScanT::TempStorage temp_storage;
    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    hipcub::CastOp<bool> cast_to_bool;
    int num = 0, nonzero = 0;
    if (index < length && cast_to_bool(inputs[index])) {
        num++;
    }
    BlockScanT(temp_storage).ExclusiveSum(num, nonzero);
    if (index < length && cast_to_bool(inputs[index])) {
        int offset = blockIdx.x ? cumulativeCounts[blockIdx.x - 1] : 0;
        output[offset + nonzero] = index;
    }
}

template<typename T>
__global__ void gatherKernel(const T *inputs, const int *indices, T *outputs, int64_t length, int step) {
    CUDA_1D_KERNEL_LOOP(index, length) {
        for (int i = 0; i < step; i++) {
            outputs[index * step + i] = inputs[indices[index] * step + i];
        }
    }
}

template<typename T>
int devicePrint(const T *deviceValues, int length, const std::string &info, int step) {
    T *values = (T *)malloc(sizeof(T) * length);
    hipMemcpy(values, deviceValues, sizeof(T) * length, hipMemcpyDeviceToHost);
    std::cout << info << ": ";
    for (int i = 0; i < length; i++) {
        if (step != 1) {
            if (!(i % step)) {
                std::cout << std::endl;
            }
        }
        std::cout << values[i] << " ";
    }
    std::cout << std::endl;
    free(values);
    return 0;
}

// Apply nonzero opeartion to scores, gather scores and boxes with returned nonzero indices.
hipError_t nonzero(float *scores, float *boxes, int length) {
    // inputs: copy scores and boxes to device
    float *deviceScores, *deviceBoxes;
    hipMalloc(&deviceScores, sizeof(float) * length);
    hipMalloc(&deviceBoxes, sizeof(float) * length * 4);
    hipMemcpy(deviceScores, scores, sizeof(float) * length, hipMemcpyHostToDevice);
    hipMemcpy(deviceBoxes, boxes, sizeof(float) * length * 4, hipMemcpyHostToDevice);

    // malloc space for nonzero block counts & cumulative counts
    int blockNum = ATenCeilDiv(length, RUNTIME_BLOCK_THREADS);
    int *devicesCounts, *devicesCumulativeCounts;
    hipMalloc(&devicesCounts, sizeof(int) * blockNum);
    hipMalloc(&devicesCumulativeCounts, sizeof(int) * blockNum);

    // count number of nonzero elements by block
    dim3 grid(blockNum);
    dim3 block(RUNTIME_BLOCK_THREADS);
    nonZeroCountKernel<float, RUNTIME_BLOCK_THREADS><<<grid, block>>>(deviceScores, (int64_t)length, devicesCounts);
    devicePrint(devicesCounts, blockNum, std::string("counts"), 1);

    // inclusive sum number of nonzero elements for each block
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, devicesCounts, devicesCumulativeCounts, blockNum);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, devicesCounts, devicesCumulativeCounts, blockNum);
    devicePrint(devicesCumulativeCounts, blockNum, std::string("cumulativeCounts"), 1);

    // retrieve nonzero indices
    int nonzeroCount, *nonzeroIndex;
    hipMemcpy(&nonzeroCount, devicesCumulativeCounts + blockNum - 1, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "nonzeroCount: " << nonzeroCount << std::endl;
    hipMalloc(&nonzeroIndex, sizeof(int) * nonzeroCount);
    nonZeroIndexKernel<float, RUNTIME_BLOCK_THREADS><<<grid, block>>>(
        deviceScores, (int64_t)length, devicesCumulativeCounts, nonzeroIndex);
    devicePrint(nonzeroIndex, nonzeroCount, std::string("nonzero"), 1);

    // retrieve nonzero elements
    float *outputScores, *outputBoxes;
    hipMalloc(&outputScores, sizeof(float) * nonzeroCount);
    hipMalloc(&outputBoxes, sizeof(float) * nonzeroCount * 4);
    gatherKernel<<<grid, block>>>(deviceScores, nonzeroIndex, outputScores, nonzeroCount, 1);
    gatherKernel<<<grid, block>>>(deviceBoxes, nonzeroIndex, outputBoxes, nonzeroCount, 4);
    devicePrint(outputScores, nonzeroCount, std::string("scores"), 1);
    devicePrint(outputBoxes, nonzeroCount * 4, std::string("boxes"), 4);

    hipFree(deviceScores);
    hipFree(deviceBoxes);
    hipFree(devicesCounts);
    hipFree(devicesCumulativeCounts);
    hipFree(d_temp_storage);
    hipFree(nonzeroIndex);
    hipFree(outputScores);
    hipFree(outputBoxes);
    return hipGetLastError();
}
